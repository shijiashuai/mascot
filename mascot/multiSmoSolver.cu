#include "hip/hip_runtime.h"
//
// Created by ss on 16-12-14.
//

#include "multiSmoSolver.h"
#include "../svm-shared/constant.h"
#include "hip/hip_runtime.h"
#include "trainingFunction.h"
#include "../svm-shared/smoGPUHelper.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"

void MultiSmoSolver::solve() {
    initCache(CACHE_SIZE);
    int nrClass = problem.getNumOfClasses();
    //train nrClass*(nrClass-1)/2 binary models
    int k = 0;
    for (int i = 0; i < nrClass; ++i) {
        for (int j = i + 1; j < nrClass; ++j) {
            printf("training classifier with label %d and %d\n", i, j);
            SvmProblem subProblem = problem.getSubProblem(i, j);
            init4Training(subProblem);
            int maxIter = (subProblem.getNumOfSamples() > INT_MAX / ITERATION_FACTOR
                           ? INT_MAX
                           : ITERATION_FACTOR * subProblem.getNumOfSamples()) * 4;
            int numOfIter;
            for (numOfIter = 0; numOfIter < maxIter && !iterate(subProblem); numOfIter++) {
                if (numOfIter % 1000 == 0 && numOfIter != 0) {
                    std::cout << ".";
                    std::cout.flush();
                }
            }

            cout << "# of iteration: " << numOfIter << endl;
            vector<int> svIndex;
            vector<float_point> coef;
            float_point rho;
            extractModel(subProblem, svIndex, coef, rho);
            model.addBinaryModel(subProblem, svIndex,coef, rho, i, j);
            k++;
            deinit4Training();
        }
    }
}

void MultiSmoSolver::initCache(int cacheSize) {
//    gpuCache = new CLATCache(cacheSize);

}

bool MultiSmoSolver::iterate(SvmProblem &subProblem) {
    int trainingSize = subProblem.getNumOfSamples();
    GetBlockMinYiGValue << < gridSize, BLOCK_SIZE >> > (devYiGValue, devAlpha, devLabel, param.C,
            trainingSize, devBlockMin, devBlockMinGlobalKey);
    //global reducer
    GetGlobalMin << < 1, BLOCK_SIZE >> >
                         (devBlockMin, devBlockMinGlobalKey, numOfBlock, devYiGValue, NULL, devBuffer);

    //copy result back to host
    checkCudaErrors(hipMemcpy(hostBuffer, devBuffer, sizeof(float_point) * 2, hipMemcpyDeviceToHost));
    int m_nIndexofSampleOne = (int) hostBuffer[0];
    float_point fMinValue;
    fMinValue = hostBuffer[1];
    float_point *devHessianSampleRow1 = devHessianMatrixCache + getHessianRow(m_nIndexofSampleOne);

    //lock cached entry for the sample one, in case it is replaced by sample two
    gpuCache->LockCacheEntry(m_nIndexofSampleOne);

    float_point fUpSelfKernelValue = 0;
    fUpSelfKernelValue = hessianDiag[m_nIndexofSampleOne];
    //select second sample

    upValue = -fMinValue;

    //get block level min (-b_ij*b_ij/a_ij)
    GetBlockMinLowValue << < gridSize, BLOCK_SIZE >> >
                                       (devYiGValue, devAlpha, devLabel, param.C, trainingSize, devHessianDiag,
                                               devHessianSampleRow1, upValue, fUpSelfKernelValue, devBlockMin, devBlockMinGlobalKey,
                                               devBlockMinYiFValue);

    //get global min
    GetGlobalMin << < 1, BLOCK_SIZE >> >
                         (devBlockMin, devBlockMinGlobalKey,
                                 numOfBlock, devYiGValue, devHessianSampleRow1, devBuffer);

    //get global min YiFValue
    //0 is the size of dynamically allocated shared memory inside kernel
    GetGlobalMin << < 1, BLOCK_SIZE >> > (devBlockMinYiFValue, numOfBlock, devBuffer);

//	hipDeviceSynchronize();
    //copy result back to host
    checkCudaErrors(hipMemcpy(hostBuffer, devBuffer, sizeof(float_point) * 4, hipMemcpyDeviceToHost));
    int m_nIndexofSampleTwo = int(hostBuffer[0]);

    //get kernel value K(Sample1, Sample2)
    float_point fKernelValue = 0;
    float_point fMinLowValue;
    fMinLowValue = hostBuffer[1];
    fKernelValue = hostBuffer[2];


    float_point *devHessianSampleRow2 = devHessianMatrixCache + getHessianRow(m_nIndexofSampleTwo);
//	hipDeviceSynchronize();


    lowValue = -hostBuffer[3];
    //check if the problem is converged
    if (upValue + lowValue <= EPS) {
        //cout << upValue << " : " << lowValue << endl;
        //m_pGPUCache->PrintCachingStatistics();
        return true;
    }

    float_point fY1AlphaDiff, fY2AlphaDiff;
    updateTwoWeight(fMinLowValue, fMinValue, m_nIndexofSampleOne, m_nIndexofSampleTwo, fKernelValue,
                    fY1AlphaDiff, fY2AlphaDiff, subProblem.v_nLabels.data());
    float_point fAlpha1 = alpha[m_nIndexofSampleOne];
    float_point fAlpha2 = alpha[m_nIndexofSampleTwo];

    gpuCache->UnlockCacheEntry(m_nIndexofSampleOne);

    //update yiFvalue
    //copy new alpha values to device
    hostBuffer[0] = m_nIndexofSampleOne;
    hostBuffer[1] = fAlpha1;
    hostBuffer[2] = m_nIndexofSampleTwo;
    hostBuffer[3] = fAlpha2;
    checkCudaErrors(hipMemcpy(devBuffer, hostBuffer, sizeof(float_point) * 4, hipMemcpyHostToDevice));
    UpdateYiFValueKernel << < gridSize, BLOCK_SIZE >> > (devAlpha, devBuffer, devYiGValue,
            devHessianSampleRow1, devHessianSampleRow2,
            fY1AlphaDiff, fY2AlphaDiff, trainingSize);
    return false;
}

void MultiSmoSolver::init4Training(const SvmProblem &subProblem) {


    unsigned int trainingSize = subProblem.getNumOfSamples();
    checkCudaErrors(hipMalloc((void **) &devAlpha, sizeof(float_point) * trainingSize));
    alpha = vector<float_point>(trainingSize,0);

    checkCudaErrors(hipMalloc((void **) &devYiGValue, sizeof(float_point) * trainingSize));
    checkCudaErrors(hipMalloc((void **) &devLabel, sizeof(int) * trainingSize));

    checkCudaErrors(hipMemset(devAlpha, 0, sizeof(float_point) * trainingSize));
    vector<float_point> revertLabel(trainingSize);
    for (int i = 0; i < trainingSize; ++i) {
        revertLabel[i] = -subProblem.v_nLabels[i];
    }
    checkCudaErrors(hipMemcpy(devYiGValue, revertLabel.data(), sizeof(float_point) * trainingSize,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devLabel, subProblem.v_nLabels.data(), sizeof(int) * trainingSize, hipMemcpyHostToDevice));

    numOfBlock = Ceil(trainingSize, BLOCK_SIZE);
    gridSize = dim3(numOfBlock > NUM_OF_BLOCK ? NUM_OF_BLOCK : numOfBlock, Ceil(numOfBlock, NUM_OF_BLOCK));
    checkCudaErrors(hipMalloc((void **) &devBlockMin, sizeof(float_point) * numOfBlock));
    checkCudaErrors(hipMalloc((void **) &devBlockMinGlobalKey, sizeof(int) * numOfBlock));
    checkCudaErrors(hipMalloc((void **) &devBlockMinYiFValue, sizeof(float_point) * numOfBlock));
    checkCudaErrors(hipMalloc((void **) &devMinValue, sizeof(float_point)));
    checkCudaErrors(hipMalloc((void **) &devMinKey, sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &devBuffer, sizeof(float_point) * 5));

    checkCudaErrors(hipHostMalloc((void **) &hostBuffer, sizeof(float_point) * 5));

    int cacheSize = CACHE_SIZE * 1024 * 256 / trainingSize;
    gpuCache = new CLATCache(subProblem.getNumOfSamples());
    gpuCache->SetCacheSize(cacheSize);
    gpuCache->InitializeCache(cacheSize, trainingSize);
    size_t sizeOfEachRowInCache;
    checkCudaErrors(
            hipMallocPitch((void **) &devHessianMatrixCache, &sizeOfEachRowInCache, trainingSize * sizeof(float_point),
                            cacheSize));
    //temp memory for reading result to cache
    numOfElementEachRowInCache = sizeOfEachRowInCache / sizeof(float_point);
    if (numOfElementEachRowInCache != trainingSize) {
        cout << "cache memory aligned to: " << numOfElementEachRowInCache
             << "; number of the training instances is: " << trainingSize << endl;
    }
    cout << "cache size v.s. ins is " << cacheSize << " v.s. " << trainingSize << endl;

    checkCudaErrors(hipMemset(devHessianMatrixCache, 0, cacheSize * sizeOfEachRowInCache));

    hessianDiag = new float_point[trainingSize];
    checkCudaErrors(hipMalloc((void **) &devHessianDiag, sizeof(float_point) * trainingSize));
    hessianCalculator = new DeviceHessianOnFly(subProblem, param.gamma);
    hessianCalculator->GetHessianDiag("", trainingSize, hessianDiag);
    checkCudaErrors(
            hipMemcpy(devHessianDiag, hessianDiag, sizeof(float_point) * trainingSize, hipMemcpyHostToDevice));
}

void MultiSmoSolver::deinit4Training() {
    checkCudaErrors(hipFree(devAlpha));
    checkCudaErrors(hipFree(devYiGValue));
    checkCudaErrors(hipFree(devLabel));
    checkCudaErrors(hipFree(devBlockMin));
    checkCudaErrors(hipFree(devBlockMinGlobalKey));
    checkCudaErrors(hipFree(devBlockMinYiFValue));
    checkCudaErrors(hipFree(devMinValue));
    checkCudaErrors(hipFree(devMinKey));
    checkCudaErrors(hipFree(devBuffer));
    checkCudaErrors(hipHostFree(hostBuffer));
    checkCudaErrors(hipFree(devHessianMatrixCache));
    checkCudaErrors(hipFree(devHessianDiag));
    delete hessianCalculator;
    delete[] hessianDiag;
}

int MultiSmoSolver::getHessianRow(int rowIndex) {
    int cacheLocation;
    bool cacheFull = false;
    bool cacheHit = gpuCache->GetDataFromCache(rowIndex,cacheLocation,cacheFull);
    if (!cacheHit) {
        if (cacheFull)
            gpuCache->ReplaceExpired(rowIndex, cacheLocation, NULL);
        hessianCalculator->ReadRow(rowIndex, devHessianMatrixCache + cacheLocation * numOfElementEachRowInCache);
    }
    return cacheLocation * numOfElementEachRowInCache;
}

void MultiSmoSolver::updateTwoWeight(float_point fMinLowValue, float_point fMinValue, int nHessianRowOneInMatrix,
                                     int nHessianRowTwoInMatrix, float_point fKernelValue, float_point &fY1AlphaDiff,
                                     float_point &fY2AlphaDiff, const int *label) {
    //get YiGValue for sample one and two
    float_point fAlpha2 = 0;
    float_point fYiFValue2 = 0;
    fAlpha2 = alpha[nHessianRowTwoInMatrix];
    fYiFValue2 = fMinLowValue;

    //get alpha values of sample
    float_point fAlpha1 = 0;
    float_point fYiFValue1 = 0;
    fAlpha1 = alpha[nHessianRowOneInMatrix];
    fYiFValue1 = fMinValue;

    //Get K(x_up, x_up), and K(x_low, x_low)
    float_point fDiag1 = 0, fDiag2 = 0;
    fDiag1 = hessianDiag[nHessianRowOneInMatrix];
    fDiag2 = hessianDiag[nHessianRowTwoInMatrix];

    //get labels of sample one and two
    int nLabel1 = 0, nLabel2 = 0;
    nLabel1 = label[nHessianRowOneInMatrix];
    nLabel2 = label[nHessianRowTwoInMatrix];

    //compute eta
    float_point eta = fDiag1 + fDiag2 - 2 * fKernelValue;
    if (eta <= 0)
        eta = TAU;

    float_point fCost1, fCost2;
//	fCost1 = Get_C(nLabel1);
//	fCost2 = Get_C(nLabel2);
    fCost1 = fCost2 = param.C;

    //keep old yi*alphas
    fY1AlphaDiff = nLabel1 * fAlpha1;
    fY2AlphaDiff = nLabel2 * fAlpha2;

    //get new alpha values
    int nSign = nLabel2 * nLabel1;
    if (nSign < 0) {
        float_point fDelta = (-nLabel1 * fYiFValue1 - nLabel2 * fYiFValue2) / eta; //(-fYiFValue1 - fYiFValue2) / eta;
        float_point fAlphaDiff = fAlpha1 - fAlpha2;
        fAlpha1 += fDelta;
        fAlpha2 += fDelta;

        if (fAlphaDiff > 0) {
            if (fAlpha2 < 0) {
                fAlpha2 = 0;
                fAlpha1 = fAlphaDiff;
            }
        } else {
            if (fAlpha1 < 0) {
                fAlpha1 = 0;
                fAlpha2 = -fAlphaDiff;
            }
        }

        if (fAlphaDiff > fCost1 - fCost2) {
            if (fAlpha1 > fCost1) {
                fAlpha1 = fCost1;
                fAlpha2 = fCost1 - fAlphaDiff;
            }
        } else {
            if (fAlpha2 > fCost2) {
                fAlpha2 = fCost2;
                fAlpha1 = fCost2 + fAlphaDiff;
            }
        }
    } //end if nSign < 0
    else {
        float_point fDelta = (nLabel1 * fYiFValue1 - nLabel2 * fYiFValue2) / eta;
        float_point fSum = fAlpha1 + fAlpha2;
        fAlpha1 -= fDelta;
        fAlpha2 += fDelta;

        if (fSum > fCost1) {
            if (fAlpha1 > fCost1) {
                fAlpha1 = fCost1;
                fAlpha2 = fSum - fCost1;
            }
        } else {
            if (fAlpha2 < 0) {
                fAlpha2 = 0;
                fAlpha1 = fSum;
            }
        }
        if (fSum > fCost2) {
            if (fAlpha2 > fCost2) {
                fAlpha2 = fCost2;
                fAlpha1 = fSum - fCost2;
            }
        } else {
            if (fAlpha1 < 0) {
                fAlpha1 = 0;
                fAlpha2 = fSum;
            }
        }
    }//end get new alpha values

    alpha[nHessianRowOneInMatrix] = fAlpha1;
    alpha[nHessianRowTwoInMatrix] = fAlpha2;

    //get alpha difference
    fY1AlphaDiff = nLabel1 * fAlpha1 - fY1AlphaDiff; //(alpha1' - alpha1) * y1
    fY2AlphaDiff = nLabel2 * fAlpha2 - fY2AlphaDiff;
}

void MultiSmoSolver::extractModel(const SvmProblem &subProblem, vector<int> &svIndex, vector<float_point> &coef,
                                  float_point &rho) const {
    const unsigned int trainingSize = subProblem.getNumOfSamples();
    vector<float_point> alpha(trainingSize);
    const vector<int> &label = subProblem.v_nLabels;
    checkCudaErrors(hipMemcpy(alpha.data(), devAlpha, sizeof(float_point) * trainingSize, hipMemcpyDeviceToHost));
    for (int i = 0; i < trainingSize; ++i) {
        if (alpha[i] != 0) {
            coef.push_back(label[i] * alpha[i]);
            svIndex.push_back(i);
        }
    }
    rho = (lowValue - upValue) / 2;
    printf("# of SV %lu\nbias = %f\n", svIndex.size(), rho);
}

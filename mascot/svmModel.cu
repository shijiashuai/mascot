#include "hip/hip_runtime.h"
/*
   * @author: created by ss on 16-11-2.
   * @brief: multi-class svm training, prediction, svm with probability output
   *
*/

#include "svmModel.h"

#include "svmPredictor.h"
#include "../svm-shared/HessianIO/deviceHessian.h"
#include "../svm-shared/storageManager.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "trainingFunction.h"
#include "multiSmoSolver.h"
#include<map>

//#include "sigmoidTrainGPUHelper.h"

//todo move these kernel functions to a proper file
//__global__ void rbfKernel(const svm_node **samples, int numOfSamples, const svm_node **supportVectors, int numOfSVs,
//                          float_point *kernelValues, float_point gamma,
//                          const float_point *coef) {
//    int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    int sampleId = idx / numOfSVs;
//    int SVId = idx % numOfSVs;
//    if (sampleId < numOfSamples) {
//        const svm_node *sample = samples[sampleId];
//        const svm_node *supportVector = supportVectors[SVId];
//        float_point sum = 0;
//        float_point d = 0;
//        int x = 0;
//        int y = 0;
//        while (sample[x].index != -1 && supportVector[y].index != -1) {
//            if (sample[x].index == supportVector[y].index)
//                d = sample[x++].value - supportVector[y++].value;
//            else if (sample[x].index < supportVector[y].index)
//                d = sample[x++].value;
//            else
//                d = supportVector[y++].value;
//            sum += d * d;
//        }
//        while (sample[x].index != -1) {
//            sum += sample[x].value * sample[x].value;
//            x++;
//        }
//        while (supportVector[y].index != -1) {
//            sum += supportVector[y].value * supportVector[y].value;
//            y++;
//        }
//        kernelValues[idx] = coef[SVId] * exp(-gamma * sum);
//    }
//};
__global__ void rbfKernel(const float_point *sampleSelfDot, int numOfSamples,
                          const float_point *svMapSelfDot, int svMapSize,
                          float_point *kernelValues, float_point gamma) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int sampleId = idx / svMapSize;
    int SVId = idx % svMapSize;
    if (sampleId < numOfSamples) {
        float_point sampleDot = sampleSelfDot[sampleId];
        float_point svDot = svMapSelfDot[SVId];
        float_point dot = kernelValues[idx];
        kernelValues[idx] = expf(-gamma * (sampleDot + svDot - 2 * dot));
    }
};

__global__ void sumKernelValues(const float_point *kernelValues, int numOfSamples, int svMapSize, int cnr2,
                                const int *svIndex, const float_point *coef,
                                const int *start, const int *count,
                                const float_point *bias, float_point *decValues) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int sampleId = idx / cnr2;
    int modelId = idx % cnr2;
    if (sampleId < numOfSamples) {
        float_point sum = 0;
        const float_point *kernelValue = kernelValues + sampleId * svMapSize;
        int si = start[modelId];
        int ci = count[modelId];
        for (int i = 0; i < ci; ++i) {
            sum += coef[si + i] * kernelValue[svIndex[si + i]];
        }
        decValues[idx] = sum - bias[modelId];
    }
}

SvmModel::~SvmModel() {
//    checkCudaErrors(hipFree(devSVs));
    checkCudaErrors(hipFree(devCoef));
    checkCudaErrors(hipFree(devStart));
    checkCudaErrors(hipFree(devCount));
    checkCudaErrors(hipFree(devProbA));
    checkCudaErrors(hipFree(devProbB));
    checkCudaErrors(hipFree(devRho));
    checkCudaErrors(hipFree(devSVMapVal));
    checkCudaErrors(hipFree(devSVMapValSelfDot));
    checkCudaErrors(hipFree(devSVMapRowPtr));
    checkCudaErrors(hipFree(devSVMapColInd));
    checkCudaErrors(hipFree(devSVIndex));
    if (svMapCSRMat) delete svMapCSRMat;
}

unsigned int SvmModel::getK(int i, int j) const {
    return ((nrClass - 1) + (nrClass - i)) * i / 2 + j - i - 1;
}

void SvmModel::fit(const SvmProblem &problem, const SVMParam &param) {
    //reset model to fit a new SvmProblem
    nrClass = problem.getNumOfClasses();
    cnr2 = (nrClass) * (nrClass - 1) / 2;
    numOfSVs = 0;
    numOfFeatures = 0;
    coef.clear();
    rho.clear();
    probA.clear();
    probB.clear();
    svIndex.clear();
    svMap.clear();
    label.clear();
    start.clear();
    count.clear();
    probability = false;

    coef.resize(cnr2);
    rho.resize(cnr2);
    probA.resize(cnr2);
    probB.resize(cnr2);
    svIndex.resize(cnr2);

    this->param = param;
    label = problem.label;
    numOfFeatures = problem.getNumOfFeatures();

    MultiSmoSolver multiSmoSolver(problem,*this,param);
    multiSmoSolver.solve();
//    train nrClass*(nrClass-1)/2 binary models
//    int k = 0;
//    for (int i = 0; i < nrClass; ++i) {
//        for (int j = i + 1; j < nrClass; ++j) {
//            printf("training classifier with label %d and %d\n", i, j);
//            SvmProblem subProblem = problem.getSubProblem(i, j);
//            if (param.probability) {
//                SVMParam probParam = param;
//                probParam.probability = 0;
//                probParam.C = 1.0;
//                SvmModel model;
//                model.fit(subProblem, probParam);
//                vector<vector<float_point> > decValues;
//                //todo predict with cross validation
//                model.predictValues(subProblem.v_vSamples, decValues);
//                for (int l = 1; l < subProblem.v_vSamples.size(); ++l) {
//                    decValues[0].push_back(decValues[l][0]);
//                }
//                sigmoidTrain(decValues.front().data(), subProblem.getNumOfSamples(), subProblem.v_nLabels, probA[k],
//                             probB[k]);
//                probability = true;
//            }
//            svm_model binaryModel = trainBinarySVM(subProblem, param);
//            addBinaryModel(subProblem, <#initializer#>, <#initializer#>, 0, j, i);
//            k++;
//        }
//    }
    int _start = 0;
    for (int i = 0; i < cnr2; ++i) {
        start.push_back(_start);
        count.push_back(svIndex[i].size());
        _start += count[i];
    }
    transferToDevice();
}

void SvmModel::transferToDevice() {
    //convert svMap to csr matrix then copy it to device
    svMapCSRMat = new CSRMatrix(svMap,numOfFeatures);
    int nnz = svMapCSRMat->getNnz();
    checkCudaErrors(hipMalloc((void **) &devSVMapVal, sizeof(float_point) * nnz));
    checkCudaErrors(hipMalloc((void **) &devSVMapValSelfDot, sizeof(float_point) * svMapCSRMat->getNumOfSamples()));
    checkCudaErrors(hipMalloc((void **) &devSVMapRowPtr, sizeof(int) * (svMapCSRMat->getNumOfSamples() + 1)));
    checkCudaErrors(hipMalloc((void **) &devSVMapColInd, sizeof(int) * nnz));
    checkCudaErrors(
            hipMemcpy(devSVMapVal, svMapCSRMat->getCSRVal(), sizeof(float_point) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(devSVMapValSelfDot, svMapCSRMat->getCSRValSelfDot(),
                       sizeof(float_point) * svMapCSRMat->getNumOfSamples(), hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(devSVMapRowPtr, svMapCSRMat->getCSRRowPtr(), sizeof(int) * (svMapCSRMat->getNumOfSamples() + 1),
                       hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devSVMapColInd, svMapCSRMat->getCSRColInd(), sizeof(int) * nnz, hipMemcpyHostToDevice));

    //flat svIndex then copy in to device
    checkCudaErrors(hipMalloc((void **) &devSVIndex, sizeof(int) * numOfSVs));
    for (int i = 0; i < cnr2; ++i) {
        checkCudaErrors(hipMemcpy(devSVIndex + start[i], svIndex[i].data(), sizeof(int) * svIndex[i].size(),
                                   hipMemcpyHostToDevice));
    }

    checkCudaErrors(hipMalloc((void **) &devCoef, sizeof(float_point) * numOfSVs));
    checkCudaErrors(hipMalloc((void **) &devStart, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devCount, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devProbA, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devProbB, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devRho, sizeof(float_point) * cnr2));
    for (int i = 0; i < cnr2; ++i) {
        checkCudaErrors(hipMemcpy(devCoef + start[i], coef[i].data(), sizeof(float_point) * count[i],
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMemcpy(devProbA, probA.data(), sizeof(float_point) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devProbB, probB.data(), sizeof(float_point) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devStart, start.data(), sizeof(int) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devCount, count.data(), sizeof(int) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devRho, rho.data(), sizeof(float_point) * cnr2, hipMemcpyHostToDevice));
}

//void SvmModel::gpu_sigmoid_train(
//	int l, const float_point *dec_values, const float_point *labels,
//	float_point& A, float_point& B)
//{
//
//	float_point prior1, prior0 ;
//	int max_iter=100;	// Maximal number of iterations
//	float_point min_step=1e-10;	// Minimal step taken in line search
//	float_point sigma=1e-12;	// For numerically strict PD of Hessian
//	float_point eps=1e-5;
//	float_point hiTarget=(prior1+1.0)/(prior1+2.0);
//	float_point loTarget=1/(prior0+2.0);
//	float_point fApB,g1,g2,gd,stepsize;
//	float_point newA,newB,newf;
//	int iter;
//	float_point fval = 0.0;
//	// Initial Point and Initial Fun Value
//	A=0.0; B=log((prior0+1.0)/(prior1+1.0));
//
//	int blocknum=(l+THREAD_NUM-1)/THREAD_NUM;
//
//	hipStream_t stream[2];
//    for(int i = 0;i < 2;i ++)
//        hipStreamCreate(&stream[i]);
//
//	float_point *dev_prior1,*dev_prior0;
//	float_point *dev_labels,*dev_t,*dev_dec_values;
//	float_point *dev_fApB,*dev_fval,*dev_sum,*dev_d1,*dev_d2,*dev_g1,*dev_h11,*dev_h21,*dev_p,*dev_q;
//	float_point *dev_det,*dev_dA,*dev_dB,*dev_gd,*dev_newf;
//	float_point *dev_newA,*dev_newB;
//
//	checkCudaErrors(hipMalloc((void**)&dev_sum,sizeof(float_point)*blocknum));
//	checkCudaErrors(hipMalloc((void**)&dev_newA,sizeof(float_point)));
//	checkCudaErrors(hipMalloc((void**)&dev_newB,sizeof(float_point)));
//	checkCudaErrors(hipMalloc((void**)&dev_fApB,sizeof(float_point)*l));
//	checkCudaErrors(hipMalloc((void**)&dev_fval,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_labels,sizeof(float_point)*l));
//	checkCudaErrors(hipMalloc((void**)&dev_t,sizeof(float_point)*l));
//	checkCudaErrors(hipMalloc((void**)&dev_dec_values,sizeof(float_point)*l));
//	checkCudaErrors(hipMalloc((void**)&dev_p,sizeof(float_point)*l));
//	checkCudaErrors(hipMalloc((void**)&dev_q,sizeof(float_point)*l));
//	checkCudaErrors(hipMalloc((void**)&dev_d1,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_d2,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_g1,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_h11,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_h21,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_det,sizeof(float_point)));
//	checkCudaErrors(hipMalloc((void**)&dev_dA,sizeof(float_point)));
//	checkCudaErrors(hipMalloc((void**)&dev_dB,sizeof(float_point)));
//	checkCudaErrors(hipMalloc((void**)&dev_gd,sizeof(float_point)));
//	checkCudaErrors(hipMalloc((void**)&dev_newf,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_prior1,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMalloc((void**)&dev_prior0,sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//
//	checkCudaErrors(hipMemcpy(dev_labels,labels,sizeof(float_point)*l,hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(dev_dec_values,dec_values,sizeof(float_point)*l,hipMemcpyHostToDevice));
//
//	checkCudaErrors(hipMemset(dev_fval, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMemset(dev_h11, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMemset(dev_h21, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMemset(dev_d1, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMemset(dev_d2, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMemset(dev_g1, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//	checkCudaErrors(hipMemset(dev_newf, 0, sizeof(float_point)*(blocknum+1)*THREAD_NUM));
//
//	dev_getprior<<<blocknum+1,THREAD_NUM>>>(dev_labels,l,dev_prior1,dev_prior0);
//	dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_prior1,dev_sum,l);
//	dev_get_sum<<<1,1>>>(dev_sum,dev_prior1,blocknum);
//	dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_prior0,dev_sum,l);
//	dev_get_sum<<<1,1>>>(dev_sum,dev_prior0,blocknum);
//
//	checkCudaErrors(hipMemcpy(&prior1,dev_prior1,sizeof(float_point),hipMemcpyDeviceToHost));
//	checkCudaErrors(hipMemcpy(&prior0,dev_prior0,sizeof(float_point),hipMemcpyDeviceToHost));
//
//	//get fApB,fval
//	dev_getfApB_fval<<<blocknum,THREAD_NUM>>>(dev_fval,dev_labels,dev_t,dev_dec_values,dev_fApB,A,B,hiTarget,loTarget,l);
//	dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_fval,dev_sum,l);
//	dev_get_sum<<<1,1>>>(dev_sum,dev_fval,blocknum);//dev_get_fval_sum<<<1,1>>>(dev_fval);
//
//	checkCudaErrors(hipFree(dev_labels));
//	for (iter=0;iter<max_iter;iter++)
//	{
//
//		if(iter>0)
//			//update newA,newB
//			dev_getfApB<<<blocknum,THREAD_NUM>>>(l,dev_fApB,dev_dec_values,A,B);
//		//get p q
//		dev_getpq<<<blocknum,THREAD_NUM>>>(l,dev_t,dev_fApB,dev_p,dev_q,dev_d1,dev_d2,dev_h11,dev_h21,dev_g1,dev_dec_values);
//		//get h11
//		dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_h11,dev_sum,l);
//		dev_get_sum<<<1,1>>>(dev_sum,dev_h11,blocknum);
//		//get h21
//		dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_h21,dev_sum,l);
//		dev_get_sum<<<1,1>>>(dev_sum,dev_h21,blocknum);
//		//get d2\h22
//		dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_d2,dev_sum,l);
//		dev_get_sum<<<1,1>>>(dev_sum,dev_d2,blocknum);//d2[0]=h22
//		//get g1
//		dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_g1,dev_sum,l);
//		dev_get_sum<<<1,1>>>(dev_sum,dev_g1,blocknum);
//		//get d1\g2
//		dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_d1,dev_sum,l);
//		dev_get_sum<<<1,1>>>(dev_sum,dev_d1,blocknum);//d1[0]=g2
//
//		checkCudaErrors(hipMemcpy(&g1,dev_g1,sizeof(float_point),hipMemcpyDeviceToHost));
//		checkCudaErrors(hipMemcpy(&g2,dev_d1,sizeof(float_point),hipMemcpyDeviceToHost));
//		// Stopping Criteria
//		if (fabs(g1)<eps && fabs(g2)<eps)
//			break;
//
//
//		// Finding Newton direction: -inv(H') * g
//		dev_get_det<<<1,1>>>(sigma,dev_h11,dev_d2,dev_h21,dev_det);
//		//?????????????
//	    dev_getdA<<<1,1,0,stream[0]>>>(dev_dA,dev_det,dev_d2,dev_h21,dev_g1,dev_d1);
//		dev_getdB<<<1,1,0,stream[1]>>>(dev_dB,dev_det,dev_h11,dev_h21,dev_g1,dev_d1);
//		dev_getgd<<<1,1>>>(dev_gd,dev_dA,dev_dB,dev_g1,dev_d1);
//
//		stepsize = 1;		// Line Search
//
//		while (stepsize >= min_step)
//		{
//			//update newA newB
//			dev_updateAB<<<1,2>>>(dev_newA,dev_newB,A,B,stepsize,dev_dA,dev_dB);
//
//			// New function value
//			dev_getnewfApB<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(l,dev_fApB,dev_dec_values,dev_newA,dev_newB);
//			dev_getnewf<<<blocknum,THREAD_NUM>>>(l,dev_fApB,dev_t,dev_newf);
//			dev_paral_red_sum<<<blocknum,THREAD_NUM,THREAD_NUM*sizeof(float_point)>>>(dev_newf,dev_sum,l);//more block?
//			dev_get_sum<<<1,1>>>(dev_sum,dev_newf,blocknum);
//
//			// Check sufficient decrease
//			checkCudaErrors(hipMemcpy(&newf,dev_newf,sizeof(float_point),hipMemcpyDeviceToHost));
//			checkCudaErrors(hipMemcpy(&fval,dev_fval,sizeof(float_point),hipMemcpyDeviceToHost));
//			checkCudaErrors(hipMemcpy(&gd,dev_gd,sizeof(float_point),hipMemcpyDeviceToHost));
//			if (newf<fval+0.0001*(float_point)stepsize*gd)
//			{
//				hipMemcpy(&A,dev_newA,sizeof(float_point),hipMemcpyDeviceToHost);
//				hipMemcpy(&B,dev_newB,sizeof(float_point),hipMemcpyDeviceToHost);
//				fval=newf;
//				break;
//			}
//			else
//				stepsize = stepsize / 2.0;
//		}
//
//		if (stepsize < min_step)
//		{
//			info("Line search fails in two-class probability estimates\n");
//			break;
//		}
//	}
//
//	if (iter>=max_iter)
//		info("Reaching maximal iterations in two-class probability estimates\n");
//
//	checkCudaErrors(hipFree(dev_newA));
//	checkCudaErrors(hipFree(dev_newB));
//	checkCudaErrors(hipFree(dev_fApB));
//	checkCudaErrors(hipFree(dev_fval));
//	checkCudaErrors(hipFree(dev_dec_values));
//	checkCudaErrors(hipFree(dev_det));
//	checkCudaErrors(hipFree(dev_dA));
//	checkCudaErrors(hipFree(dev_dB));
//	checkCudaErrors(hipFree(dev_gd));
//	checkCudaErrors(hipFree(dev_newf));
//	checkCudaErrors(hipFree(dev_t));
//	checkCudaErrors(hipFree(dev_d1));
//	checkCudaErrors(hipFree(dev_d2));
//	checkCudaErrors(hipFree(dev_g1));
//	checkCudaErrors(hipFree(dev_h11));
//	checkCudaErrors(hipFree(dev_h21));
//	checkCudaErrors(hipFree(dev_p));
//	checkCudaErrors(hipFree(dev_q));
//	checkCudaErrors(hipFree(dev_sum));
//	checkCudaErrors(hipFree(dev_prior1));
//	checkCudaErrors(hipFree(dev_prior0));
//}


void SvmModel::sigmoidTrain(const float_point *decValues, const int l, const vector<int> &labels, float_point &A,
                            float_point &B) {
    double prior1 = 0, prior0 = 0;
    int i;

    for (i = 0; i < l; i++)
        if (labels[i] > 0)
            prior1 += 1;
        else
            prior0 += 1;

    int max_iter = 100;    // Maximal number of iterations
    double min_step = 1e-10;    // Minimal step taken in line search
    double sigma = 1e-12;    // For numerically strict PD of Hessian
    double eps = 1e-5;
    double hiTarget = (prior1 + 1.0) / (prior1 + 2.0);
    double loTarget = 1 / (prior0 + 2.0);
    double *t = (double *) malloc(sizeof(double) * l);
    double fApB, p, q, h11, h22, h21, g1, g2, det, dA, dB, gd, stepsize;
    double newA, newB, newf, d1, d2;
    int iter;

    // Initial Point and Initial Fun Value
    A = 0.0;
    B = log((prior0 + 1.0) / (prior1 + 1.0));
    double fval = 0.0;

    for (i = 0; i < l; i++) {
        if (labels[i] > 0)
            t[i] = hiTarget;
        else
            t[i] = loTarget;
        fApB = decValues[i] * A + B;
        if (fApB >= 0)
            fval += t[i] * fApB + log(1 + exp(-fApB));
        else
            fval += (t[i] - 1) * fApB + log(1 + exp(fApB));
    }
    for (iter = 0; iter < max_iter; iter++) {
        // Update Gradient and Hessian (use H' = H + sigma I)
        h11 = sigma; // numerically ensures strict PD
        h22 = sigma;
        h21 = 0.0;
        g1 = 0.0;
        g2 = 0.0;
        for (i = 0; i < l; i++) {
            fApB = decValues[i] * A + B;
            if (fApB >= 0) {
                p = exp(-fApB) / (1.0 + exp(-fApB));
                q = 1.0 / (1.0 + exp(-fApB));
            } else {
                p = 1.0 / (1.0 + exp(fApB));
                q = exp(fApB) / (1.0 + exp(fApB));
            }
            d2 = p * q;
            h11 += decValues[i] * decValues[i] * d2;
            h22 += d2;
            h21 += decValues[i] * d2;
            d1 = t[i] - p;
            g1 += decValues[i] * d1;
            g2 += d1;
        }

        // Stopping Criteria
        if (fabs(g1) < eps && fabs(g2) < eps)
            break;

        // Finding Newton direction: -inv(H') * g
        det = h11 * h22 - h21 * h21;
        dA = -(h22 * g1 - h21 * g2) / det;
        dB = -(-h21 * g1 + h11 * g2) / det;
        gd = g1 * dA + g2 * dB;

        stepsize = 1;        // Line Search
        while (stepsize >= min_step) {
            newA = A + stepsize * dA;
            newB = B + stepsize * dB;

            // New function value
            newf = 0.0;
            for (i = 0; i < l; i++) {
                fApB = decValues[i] * newA + newB;
                if (fApB >= 0)
                    newf += t[i] * fApB + log(1 + exp(-fApB));
                else
                    newf += (t[i] - 1) * fApB + log(1 + exp(fApB));
            }
            // Check sufficient decrease
            if (newf < fval + 0.0001 * stepsize * gd) {
                A = newA;
                B = newB;
                fval = newf;
                break;
            } else
                stepsize = stepsize / 2.0;
        }

        if (stepsize < min_step) {
            printf("Line search fails in two-class probability estimates\n");
            break;
        }
    }

    if (iter >= max_iter)
        printf(
                "Reaching maximal iterations in two-class probability estimates\n");
    free(t);
}

void SvmModel::addBinaryModel(const SvmProblem &problem, const vector<int> &svIndex, const vector<float_point> &coef,
                              float_point rho, int i,
                              int j) {
    static map<int, int> indexMap;
    int k = getK(i, j);
    this->coef[k] = coef;
    for (int l = 0; l < svIndex.size(); ++l) {
        int originalIndex = problem.originalIndex[svIndex[l]];
        if (indexMap.find(originalIndex) != indexMap.end()) {
        } else {
            indexMap[originalIndex] = svMap.size();
            svMap.push_back(problem.v_vSamples[svIndex[l]]);
        }
        this->svIndex[k].push_back(indexMap[originalIndex]);
    }
    this->rho[k] = rho;
    numOfSVs += svIndex.size();
}

void
SvmModel::predictValues(const vector<vector<svm_node> > &v_vSamples,
                        vector<vector<float_point> > &decisionValues) const {
    //copy samples to device
    CSRMatrix sampleCSRMat(v_vSamples, numOfFeatures);
    float_point *devSampleVal;
    float_point *devSampleValSelfDot;
    int *devSampleRowPtr;
    int *devSampleColInd;
    int sampleNnz = sampleCSRMat.getNnz();
    checkCudaErrors(hipMalloc((void **) &devSampleVal, sizeof(float_point) * sampleNnz));
    checkCudaErrors(hipMalloc((void **) &devSampleValSelfDot, sizeof(float_point) * sampleCSRMat.getNumOfSamples()));
    checkCudaErrors(hipMalloc((void **) &devSampleRowPtr, sizeof(int) * (sampleCSRMat.getNumOfSamples() + 1)));
    checkCudaErrors(hipMalloc((void **) &devSampleColInd, sizeof(int) * sampleNnz));
    checkCudaErrors(hipMemcpy(devSampleVal, sampleCSRMat.getCSRVal(), sizeof(float_point) * sampleNnz,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devSampleValSelfDot, sampleCSRMat.getCSRValSelfDot(),
                               sizeof(float_point) * sampleCSRMat.getNumOfSamples(), hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(devSampleRowPtr, sampleCSRMat.getCSRRowPtr(), sizeof(int) * (sampleCSRMat.getNumOfSamples() + 1),
                       hipMemcpyHostToDevice));
    checkCudaErrors(
            hipMemcpy(devSampleColInd, sampleCSRMat.getCSRColInd(), sizeof(int) * sampleNnz, hipMemcpyHostToDevice));

    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    float_point *devKernelValues;
    checkCudaErrors(hipMalloc((void **) &devKernelValues,
                               sizeof(float_point) * v_vSamples.size() * svMap.size()));

    //dot product between sv and sample
    CSRMatrix::CSRmm2Dense(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                           sampleCSRMat.getNumOfSamples(), svMapCSRMat->getNumOfSamples(),
                           svMapCSRMat->getNumOfFeatures(),
                           descr, sampleNnz, devSampleVal, devSampleRowPtr, devSampleColInd,
                           descr, svMapCSRMat->getNnz(), devSVMapVal, devSVMapRowPtr, devSVMapColInd,
                           devKernelValues);

    //obtain exp(-gamma*(a^2+b^2-2ab))
    int numOfBlock = Ceil(v_vSamples.size() * svMap.size(), BLOCK_SIZE);
    rbfKernel << < numOfBlock, BLOCK_SIZE >> >
                               (devSampleValSelfDot, sampleCSRMat.getNumOfSamples(),
                                       devSVMapValSelfDot, svMapCSRMat->getNumOfSamples(),
                                       devKernelValues, param.gamma);

    //sum kernel values of each model then obtain decision values
    numOfBlock = Ceil(v_vSamples.size() * cnr2, BLOCK_SIZE);
    float_point *devDecisionValues;
    checkCudaErrors(hipMalloc((void **) &devDecisionValues, sizeof(float_point) * v_vSamples.size() * cnr2));
    sumKernelValues << < numOfBlock, BLOCK_SIZE >> > (devKernelValues, v_vSamples.size(),
            svMapCSRMat->getNumOfSamples(), cnr2, devSVIndex, devCoef, devStart, devCount, devRho, devDecisionValues);
    float_point *tempDecValues = new float_point[v_vSamples.size() * cnr2];
    checkCudaErrors(hipMemcpy(tempDecValues, devDecisionValues,
                               sizeof(float_point) * v_vSamples.size() * cnr2, hipMemcpyDeviceToHost));
    decisionValues = vector<vector<float_point> >(v_vSamples.size(), vector<float_point>(cnr2));
    for (int i = 0; i < decisionValues.size(); ++i) {
        memcpy(decisionValues[i].data(), tempDecValues + i * cnr2, sizeof(float_point) * cnr2);
    }
    delete[] tempDecValues;
    checkCudaErrors(hipFree(devDecisionValues));
    checkCudaErrors(hipFree(devKernelValues));
    checkCudaErrors(hipFree(devSampleVal));
    checkCudaErrors(hipFree(devSampleValSelfDot));
    checkCudaErrors(hipFree(devSampleRowPtr));
    checkCudaErrors(hipFree(devSampleColInd));
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
}

vector<int> SvmModel::predict(const vector<vector<svm_node> > &v_vSamples, bool probability) const {
    vector<int> labels;
    if (!probability) {
        vector<vector<float_point> > decisionValues;
        predictValues(v_vSamples, decisionValues);
        for (int l = 0; l < v_vSamples.size(); ++l) {
            vector<int> votes(nrClass, 0);
            int k = 0;
            for (int i = 0; i < nrClass; ++i) {
                for (int j = i + 1; j < nrClass; ++j) {
                    if (decisionValues[l][k++] > 0)
                        votes[i]++;
                    else
                        votes[j]++;
                }
            }
            int maxVoteClass = 0;
            for (int i = 0; i < nrClass; ++i) {
                if (votes[i] > votes[maxVoteClass])
                    maxVoteClass = i;
            }
            labels.push_back(this->label[maxVoteClass]);
        }
    } else {
        printf("predict with probability\n");
        assert(this->probability);
        vector<vector<float_point> > prob = predictProbability(v_vSamples);
        // todo select max using GPU
        for (int i = 0; i < v_vSamples.size(); ++i) {
            int maxProbClass = 0;
            for (int j = 0; j < nrClass; ++j) {
                if (prob[i][j] > prob[i][maxProbClass])
                    maxProbClass = j;
            }
            labels.push_back(this->label[maxProbClass]);
        }
    }
    return labels;
}

float_point SvmModel::sigmoidPredict(float_point decValue, float_point A, float_point B) const {
    double fApB = decValue * A + B;
    // 1-p used later; avoid catastrophic cancellation
    if (fApB >= 0)
        return exp(-fApB) / (1.0 + exp(-fApB));
    else
        return 1.0 / (1 + exp(fApB));
}

void SvmModel::multiClassProbability(const vector<vector<float_point> > &r, vector<float_point> &p) const {
    int t, j;
    int iter = 0, max_iter = max(100, nrClass);
    double **Q = (double **) malloc(sizeof(double *) * nrClass);
    double *Qp = (double *) malloc(sizeof(double) * nrClass);
    double pQp, eps = 0.005 / nrClass;

    for (t = 0; t < nrClass; t++) {
        p[t] = 1.0 / nrClass;  // Valid if k = 1
        Q[t] = (double *) malloc(sizeof(double) * nrClass);
        Q[t][t] = 0;
        for (j = 0; j < t; j++) {
            Q[t][t] += r[j][t] * r[j][t];
            Q[t][j] = Q[j][t];
        }
        for (j = t + 1; j < nrClass; j++) {
            Q[t][t] += r[j][t] * r[j][t];
            Q[t][j] = -r[j][t] * r[t][j];
        }
    }
    for (iter = 0; iter < max_iter; iter++) {
        // stopping condition, recalculate QP,pQP for numerical accuracy
        pQp = 0;
        for (t = 0; t < nrClass; t++) {
            Qp[t] = 0;
            for (j = 0; j < nrClass; j++)
                Qp[t] += Q[t][j] * p[j];
            pQp += p[t] * Qp[t];
        }
        double max_error = 0;
        for (t = 0; t < nrClass; t++) {
            double error = fabs(Qp[t] - pQp);
            if (error > max_error)
                max_error = error;
        }
        if (max_error < eps)
            break;

        for (t = 0; t < nrClass; t++) {
            double diff = (-Qp[t] + pQp) / Q[t][t];
            p[t] += diff;
            pQp = (pQp + diff * (diff * Q[t][t] + 2 * Qp[t])) / (1 + diff)
                  / (1 + diff);
            for (j = 0; j < nrClass; j++) {
                Qp[j] = (Qp[j] + diff * Q[t][j]) / (1 + diff);
                p[j] /= (1 + diff);
            }
        }
    }
    if (iter >= max_iter)
        printf("Exceeds max_iter in multiclass_prob\n");
    for (t = 0; t < nrClass; t++)
        free(Q[t]);
    free(Q);
    free(Qp);
}

vector<vector<float_point> > SvmModel::predictProbability(const vector<vector<svm_node> > &v_vSamples) const {
    vector<vector<float_point> > result;
    vector<vector<float_point> > decValues;
    predictValues(v_vSamples, decValues);
    for (int l = 0; l < v_vSamples.size(); ++l) {
        vector<vector<float_point> > r(nrClass, vector<float_point>(nrClass));
        double min_prob = 1e-7;
        int k = 0;
        for (int i = 0; i < nrClass; i++)
            for (int j = i + 1; j < nrClass; j++) {
                r[i][j] = min(
                        max(sigmoidPredict(decValues[l][k], probA[k], probB[k]), min_prob), 1 - min_prob);
                r[j][i] = 1 - r[i][j];
                k++;
            }
        vector<float_point> p(nrClass);
        multiClassProbability(r, p);
        result.push_back(p);
    }
    return result;
}

bool SvmModel::isProbability() const {
    return probability;
}


